#include "hip/hip_runtime.h"
// Probabilistic Question-Answering system
// @2017 Sarge Rogatch
// This software is distributed under GNU AGPLv3 license. See file LICENSE in repository root for details.

#include <cstdint>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <>
#include "../PqaCore/CudaEngineGpu.cuh"

namespace ProbQA {

template<typename taNumber> class DevAccumulator {
  taNumber _sum, _corr;
public:
  __device__ DevAccumulator& Init(const taNumber value) {
    _sum = value;
    _corr = 0;
    return *this;
  }

  __device__ DevAccumulator& Add(const taNumber value) {
    const taNumber y = value - _corr;
    const taNumber t = _sum + y;
    _corr = (t - _sum) - y;
    _sum = t;
    return *this;
  }

  __device__ DevAccumulator& Add(const DevAccumulator<taNumber>& fellow) {
    return Add(fellow._sum).Add(fellow._corr);
  }

  __device__ taNumber Get() {
    return _sum;
  }
};

__device__ bool TestBit(const uint8_t *pArr, const int64_t iBit) {
  return pArr[iBit >> 3] & (1 << (iBit & 7));
}

template<typename taNumber> __device__ taNumber& GetSA(const int64_t iQuestion, const int64_t iAnswer,
  const int64_t iTarget, taNumber *pSA, const int64_t nAnswers, const int64_t nTargets)
{
  return pSA[(iQuestion * nAnswers + iAnswer) * nTargets + iTarget];
}

template<typename taNumber> __device__ taNumber& GetMD(const int64_t iQuestion, const int64_t iTarget,
  taNumber *pMD, const int64_t nTargets)
{
  return pMD[iQuestion * nTargets + iTarget];
}

template<typename taNumber> __global__ void InitStatistics(const InitStatisticsKernel<taNumber> isk) {
  const int32_t nThreads = gridDim.x * blockDim.x;
  int64_t iInstance = threadIdx.x + blockIdx.x * blockDim.x;
  while (iInstance < isk._nSAItems) {
    isk._psA[iInstance] = isk._initSqr;
    iInstance += nThreads;
  }

  iInstance = threadIdx.x + blockIdx.x * blockDim.x;
  while (iInstance < isk._nMDItems) {
    isk._pmD[iInstance] = isk._initMD;
    iInstance += nThreads;
  }

  iInstance = threadIdx.x + blockIdx.x * blockDim.x;
  while (iInstance < isk._nVBItems) {
    isk._pvB[iInstance] = isk._init1;
    iInstance += nThreads;
  }
}

template<typename taNumber> void InitStatisticsKernel<taNumber>::Run(const KernelLaunchContext& klc,
  hipStream_t stream)
{
  const uint32_t nBlocks = klc.GetBlockCount(_nSAItems);
  InitStatistics<taNumber> << <nBlocks, klc.DefaultBlockSize(), /* no shared memory */ 0, stream >> > (*this);
}


template<typename taNumber> __global__ void StartQuiz(const StartQuizKernel<taNumber> sqk) {
  extern __shared__ DevAccumulator<taNumber> sum[];
  int64_t iInstance = threadIdx.x;
  sum[iInstance].Init((iInstance < sqk._nTargets && !TestBit(sqk._pTargetGaps, iInstance)) ? sqk._pvB[iInstance] : 0);
  for (;;) {
    iInstance += blockDim.x;
    if (iInstance >= sqk._nTargets) {
      break;
    }
    if (!TestBit(sqk._pTargetGaps, iInstance)) {
      sum[threadIdx.x].Add(sqk._pvB[iInstance]);
    }
  }
  __syncthreads();
  uint32_t remains = blockDim.x >> 1;
  for (; remains > KernelLaunchContext::_cWarpSize; remains >>= 1) {
    if (threadIdx.x < remains) {
      sum[threadIdx.x].Add(sum[threadIdx.x + remains]);
    }
    __syncthreads();
  }
  if (threadIdx.x < KernelLaunchContext::_cWarpSize) {
    for (; remains >= 1; remains >>= 1) {
      if (threadIdx.x < remains) {
        sum[threadIdx.x].Add(sum[threadIdx.x + remains]);
      }
    }
  }
  __syncthreads();
  const taNumber divisor = sum[0].Get();
  const taNumber multiplier = 1 / divisor;
  iInstance = threadIdx.x;
  while (iInstance < sqk._nTargets) {
    sqk._pPriorMants[iInstance] = (TestBit(sqk._pTargetGaps, iInstance) ? 0 : sqk._pvB[iInstance] * multiplier);
    iInstance += blockDim.x;
  }

  const int64_t nQAskedComps = (sqk._nTargets + 63) >> 6;
  iInstance = threadIdx.x;
  while (iInstance < nQAskedComps) {
    reinterpret_cast<uint64_t*>(sqk._pQAsked)[iInstance] = 0;
    iInstance += blockDim.x;
  }
}

template<typename taNumber> void StartQuizKernel<taNumber>::Run(const KernelLaunchContext& klc, hipStream_t stream) {
  const uint32_t nThreads = klc.FixBlockSize(_nTargets);
  StartQuiz<taNumber> << <1, nThreads, sizeof(DevAccumulator<taNumber>) * nThreads, stream >> > (*this);
}

template<typename taNumber> struct EvaluateQuestionShared {
  DevAccumulator<taNumber> _accLhEnt;
  DevAccumulator<taNumber> _accLack;
  DevAccumulator<taNumber> _accVelocity;
};

template<typename taNumber> __device__ void EvaluateQuestion(const int64_t iQuestion,
  const NextQuestionKernel<taNumber>& nqk, EvaluateQuestionShared<taNumber> *shared)
{
  // i - questions
  // k - answers
  // j - targets
  __shared__ DevAccumulator<taNumber> accTotW;
  if (threadIdx.x == 0) {
    accTotW.Init(0);
  }
  uint32_t remains;
  for (int64_t iAnswer = 0; iAnswer < nqk._nAnswers; iAnswer++) {
    const bool isAns0 = (iAnswer == 0);
    shared[threadIdx.x]._accLhEnt.Init(0);
    shared[threadIdx.x]._accLack.Init(0);
    shared[threadIdx.x]._accVelocity.Init(0);
    for (int64_t blockFirst = 0; blockFirst < nqk._nTargets; blockFirst += blockDim.x) {
      const int64_t iTarget = threadIdx.x + blockFirst;
      if (iTarget < nqk._nTargets) {
        taNumber postLikelihood;
        if (TestBit(nqk._pTargetGaps, iTarget)) {
          postLikelihood = 0;
        }
        else {
          taNumber invCountTotal;
          if (isAns0) {
            invCountTotal = 1 / GetMD(iQuestion, iTarget, nqk._pmD, nqk._nTargets);
            nqk._pInvD[blockIdx.x*nqk._nTargets + iTarget] = invCountTotal;
          }
          else {
            invCountTotal = nqk._pInvD[blockIdx.x*nqk._nTargets + iTarget];
          }
          const taNumber Pr_Qi_eq_k_given_Tj = GetSA(iQuestion, iAnswer, iTarget, nqk._psA, nqk._nAnswers,
            nqk._nTargets) * invCountTotal;
          postLikelihood = Pr_Qi_eq_k_given_Tj * nqk._pPriorMants[iTarget];
          shared[threadIdx.x]._accLhEnt.Add(postLikelihood);
        }
        nqk._pPosteriors[blockIdx.x*nqk._nTargets + iTarget] = postLikelihood;
      }
    }
    __syncthreads(); // get the shared data in all threads
    remains = blockDim.x >> 1;
    for (; remains > KernelLaunchContext::_cWarpSize; remains >>= 1) {
      if (threadIdx.x < remains) {
        shared[threadIdx.x]._accLhEnt.Add(shared[threadIdx.x + remains]._accLhEnt);
      }
      __syncthreads();
    }
    if (threadIdx.x < KernelLaunchContext::_cWarpSize) {
      for (; remains >= 1; remains >>= 1) {
        if (threadIdx.x < remains) {
          shared[threadIdx.x]._accLhEnt.Add(shared[threadIdx.x + remains]._accLhEnt);
        }
      }
    }
    __syncthreads(); // Ensure that all threads get updated shared[0]
    const taNumber Wk = shared[0]._accLhEnt.Get();
    if (threadIdx.x == 0) {
      accTotW.Add(Wk);
      nqk._pAnsMets[blockIdx.x*nqk._nAnswers + iAnswer]._weight = Wk;
    }
    __syncthreads(); // Ensure that all threads no more need shared[0]

    const taNumber invWk = 1 / Wk;
    shared[threadIdx.x]._accLhEnt.Init(0); // reuse for entropy summation

    for (int64_t blockFirst = 0; blockFirst < nqk._nTargets; blockFirst += blockDim.x) {
      const int64_t iTarget = threadIdx.x + blockFirst;
      if (iTarget < nqk._nTargets && !TestBit(nqk._pTargetGaps, iTarget)) {
        const taNumber posterior = nqk._pPosteriors[blockIdx.x*nqk._nTargets + iTarget] * invWk;
        const taNumber prior = nqk._pPriorMants[iTarget];
        taNumber Hikj, lack, l2post;
        if (posterior == 0 || (l2post = log2(posterior)) == 0) {
          Hikj = 0;
          lack = 0;
        }
        else {
          Hikj = l2post * posterior;
          //const taNumber invDij = nqk._pInvD[blockIdx.x*nqk._nTargets + iTarget];
          //lack = invDij * invDij / l2post;
          const taNumber invSikj = 1 / GetSA(iQuestion, iAnswer, iTarget, nqk._psA, nqk._nAnswers,
            nqk._nTargets);
          lack = invSikj * invSikj / l2post;
        }
        shared[threadIdx.x]._accLhEnt.Add(Hikj);
        shared[threadIdx.x]._accLack.Add(lack);

        const taNumber diff = posterior - prior;
        const taNumber square = diff * diff;
        shared[threadIdx.x]._accVelocity.Add(square);
      }
    }
    __syncthreads(); // get the shared data in all threads
    remains = blockDim.x >> 1;
    for (; remains > KernelLaunchContext::_cWarpSize; remains >>= 1) {
      if (threadIdx.x < remains) {
        shared[threadIdx.x]._accLhEnt.Add(shared[threadIdx.x + remains]._accLhEnt);
        shared[threadIdx.x]._accLack.Add(shared[threadIdx.x + remains]._accLack);
        shared[threadIdx.x]._accVelocity.Add(shared[threadIdx.x + remains]._accVelocity);
      }
      __syncthreads();
    }
    if (threadIdx.x < KernelLaunchContext::_cWarpSize) {
      for (; remains >= 1; remains >>= 1) {
        if (threadIdx.x < remains) {
          shared[threadIdx.x]._accLhEnt.Add(shared[threadIdx.x + remains]._accLhEnt);
          shared[threadIdx.x]._accLack.Add(shared[threadIdx.x + remains]._accLack);
          shared[threadIdx.x]._accVelocity.Add(shared[threadIdx.x + remains]._accVelocity);
        }
      }
      if (threadIdx.x == 0) {
        const int64_t iAnsMet = blockIdx.x*nqk._nAnswers + iAnswer;
        nqk._pAnsMets[iAnsMet]._entropy = shared[0]._accLhEnt.Get();
        nqk._pAnsMets[iAnsMet]._lack = shared[0]._accLack.Get();
        nqk._pAnsMets[iAnsMet]._velocity = sqrt(shared[0]._accVelocity.Get());
      }
    }
  }

  shared[threadIdx.x]._accLhEnt.Init(0);
  shared[threadIdx.x]._accLack.Init(0);
  shared[threadIdx.x]._accVelocity.Init(0);
  int64_t iAnswer = threadIdx.x;
  while (iAnswer < nqk._nAnswers) {
    const int64_t iAnsMet = blockIdx.x*nqk._nAnswers + iAnswer;
    const taNumber weight = nqk._pAnsMets[iAnsMet]._weight;
    shared[threadIdx.x]._accLhEnt.Add(nqk._pAnsMets[iAnsMet]._entropy * weight);
    shared[threadIdx.x]._accLack.Add(nqk._pAnsMets[iAnsMet]._lack * weight);
    shared[threadIdx.x]._accVelocity.Add(nqk._pAnsMets[iAnsMet]._velocity * weight);
    iAnswer += blockDim.x;
  }
  __syncthreads();
  remains = blockDim.x >> 1;
  for (; remains > KernelLaunchContext::_cWarpSize; remains >>= 1) {
    if (threadIdx.x < remains) {
      shared[threadIdx.x]._accLhEnt.Add(shared[threadIdx.x + remains]._accLhEnt);
      shared[threadIdx.x]._accLack.Add(shared[threadIdx.x + remains]._accLack);
      shared[threadIdx.x]._accVelocity.Add(shared[threadIdx.x + remains]._accVelocity);
    }
    __syncthreads();
  }
  if (threadIdx.x < KernelLaunchContext::_cWarpSize) {
    for (; remains >= 1; remains >>= 1) {
      if (threadIdx.x < remains) {
        shared[threadIdx.x]._accLhEnt.Add(shared[threadIdx.x + remains]._accLhEnt);
        shared[threadIdx.x]._accLack.Add(shared[threadIdx.x + remains]._accLack);
        shared[threadIdx.x]._accVelocity.Add(shared[threadIdx.x + remains]._accVelocity);
      }
    }
    if (threadIdx.x == 0) {
      const taNumber totW = accTotW.Get(); // actually this must be equal to 1 (+-)
      const taNumber normalizer = 1 / totW;
      const taNumber avgH = -shared[0]._accLhEnt.Get() * normalizer;
      const taNumber avgL = -shared[0]._accLack.Get() * normalizer;
      const taNumber avgV = shared[0]._accVelocity.Get() * normalizer;
      const taNumber nExpectedTargets = exp2(avgH);
      nqk._pTotals[iQuestion] = pow(avgL, 1) * pow(avgV + 1e-4, 9) * pow(nExpectedTargets, -2);
    }
  }
}

template<typename taNumber> __global__ void NextQuestion(const NextQuestionKernel<taNumber> nqk) {
  extern __shared__ EvaluateQuestionShared<taNumber> shared[];
//  __shared__ int64_t prevTime;
  int64_t iQuestion = blockIdx.x;
  while (iQuestion < nqk._nQuestions) {
    if ( TestBit(nqk._pQAsked, iQuestion) || TestBit(nqk._pQuestionGaps, iQuestion) ) {
      if (threadIdx.x == 0) {
        nqk._pTotals[iQuestion] = 0;
      }
    }
    else {
      EvaluateQuestion(iQuestion, nqk, shared);
    }
    iQuestion += gridDim.x;
  }
}

template<typename taNumber> void NextQuestionKernel<taNumber>::Run(hipStream_t stream)
{
  NextQuestion<taNumber><<<_nBlocks, _nThreadsPerBlock, sizeof(EvaluateQuestionShared<taNumber>) * _nThreadsPerBlock,
    stream>>>(*this);
}


template<typename taNumber> __device__ taNumber GetUpdatedPrior(const RecordAnswerKernel<taNumber>& rak,
  const int64_t iTarget)
{
  return rak._pPriorMants[iTarget] * GetSA(rak._iQuestion, rak._iAnswer, iTarget, rak._psA, rak._nAnswers,
    rak._nTargets) / GetMD(rak._iQuestion, iTarget, rak._pmD, rak._nTargets);
}

template<typename taNumber> __global__ void RecordAnswer(RecordAnswerKernel<taNumber> rak) {
  extern __shared__ DevAccumulator<taNumber> sum[];
  int64_t iTarget = threadIdx.x;
  if (iTarget < rak._nTargets) {
    taNumber updPrior = GetUpdatedPrior(rak, iTarget);
    sum[iTarget].Init(updPrior);
    rak._pPriorMants[iTarget] = updPrior;
    for (;;) {
      iTarget += blockDim.x;
      if (iTarget >= rak._nTargets) {
        break;
      }
      updPrior = GetUpdatedPrior(rak, iTarget);
      sum[iTarget].Add(updPrior);
      rak._pPriorMants[iTarget] = updPrior;
    }
  }
  else {
    sum[iTarget].Init(0);
  }
  __syncthreads();
  uint32_t remains = blockDim.x >> 1;
  for (; remains > KernelLaunchContext::_cWarpSize; remains >>= 1) {
    if (threadIdx.x < remains) {
      sum[threadIdx.x].Add(sum[threadIdx.x + remains]);
    }
    __syncthreads();
  }
  if (threadIdx.x < KernelLaunchContext::_cWarpSize) {
    for (; remains >= 1; remains >>= 1) {
      if (threadIdx.x < remains) {
        sum[threadIdx.x].Add(sum[threadIdx.x + remains]);
      }
    }
  }
  __syncthreads();
  const taNumber normalizer = 1 / sum[0].Get();
  iTarget = threadIdx.x;
  while (iTarget < rak._nTargets) {
    rak._pPriorMants[iTarget] *= normalizer;
    iTarget += blockDim.x;
  }
}

template<typename taNumber> void RecordAnswerKernel<taNumber>::Run(const KernelLaunchContext& klc, hipStream_t stream)
{
  const uint32_t nThreads = klc.FixBlockSize(_nTargets);
  RecordAnswer<taNumber><<<1, nThreads, sizeof(DevAccumulator<taNumber>)*nThreads, stream>>>(*this);
}


template<typename taNumber> __global__ void RecordQuizTarget(const RecordQuizTargetKernel<taNumber> rqtk) {
  const uint32_t nThreads = blockDim.x * gridDim.x;
  int64_t iAQ = threadIdx.x + blockIdx.x*blockDim.x;
  if (iAQ == 0) {
    rqtk._pvB[rqtk._iTarget] += rqtk._amount;
  }
  while (iAQ < rqtk._nAQs) {
    const int64_t iQuestion = rqtk._pAQs[iAQ]._iQuestion;
    if (iAQ == 0 || iQuestion != rqtk._pAQs[iAQ - 1]._iQuestion) {
      int64_t iSame = iAQ;
      do {
        const int64_t iAnswer = rqtk._pAQs[iSame]._iAnswer;
        taNumber& Aikj = GetSA(iQuestion, iAnswer, rqtk._iTarget, rqtk._psA, rqtk._nAnswers, rqtk._nTargets);;
        const taNumber aSquare = Aikj;
        const taNumber a = sqrt(aSquare);
        const taNumber addend = a * rqtk._twoB + rqtk._bSquare;
        Aikj += addend;
        GetMD(iQuestion, rqtk._iTarget, rqtk._pmD, rqtk._nTargets) += addend;
        iSame++;
      } while (iSame < rqtk._nAQs && rqtk._pAQs[iSame]._iQuestion == iQuestion);
    }
    iAQ += nThreads;
  }
}

template<typename taNumber> void RecordQuizTargetKernel<taNumber>::Run(const KernelLaunchContext& klc,
  hipStream_t stream)
{
  const uint32_t nThreads = klc.FixBlockSize(_nTargets);
  const uint32_t nBlocks = klc.GetBlockCount(_nAQs, nThreads);
  RecordQuizTarget<taNumber><<<nBlocks, nThreads, /* no shared memory */ 0, stream>>>(*this);
}

//// Instantinations
template struct InitStatisticsKernel<float>;
template struct StartQuizKernel<float>;
template struct NextQuestionKernel<float>;
template struct RecordAnswerKernel<float>;
template struct RecordQuizTargetKernel<float>;

} // namespace ProbQA
